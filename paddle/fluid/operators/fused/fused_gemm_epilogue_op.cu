#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.
Copyright (c) 2022 NVIDIA Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <algorithm>
#include <mutex>
#include <unordered_map>
#include "paddle/fluid/platform/float16.h"

#include "gflags/gflags.h"
#include "glog/logging.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/framework/op_version_registry.h"
#include "paddle/phi/backends/all_context.h"
#include "paddle/phi/backends/gpu/rocm/rocm_helper.h"
#include "paddle/phi/common/amp_type_traits.h"
#include "paddle/phi/common/float16.h"
#include "paddle/phi/common/memory_utils.h"
#include "paddle/phi/core/dense_tensor.h"
#include "paddle/phi/core/enforce.h"
#include "paddle/phi/core/scope_guard.h"
#include "paddle/phi/kernels/funcs/blas/blas.h"
#include "paddle/phi/kernels/funcs/fc_functor.h"
#include "paddle/phi/kernels/gpu/reduce.h"
#include "paddle/phi/kernels/primitive/functor_primitives.h"
#include "paddle/utils/optional.h"

namespace paddle {
namespace operators {

template <typename T>
struct FcTypeTraits;

template <>
struct FcTypeTraits<float> {
  typedef float4 Type;
};

template <>
struct FcTypeTraits<double> {
  typedef double4 Type;
};

struct float16_4 {
  float16 x, y, z, w;
};

template <>
struct FcTypeTraits<float16> {
  typedef float16_4 Type;
};

template <typename T, bool DoRelu>
__global__ void bias_relu_v4(const int num, const T* bias, T* data, int K) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num) {
    int bias_idx = tid % K;
    const T bias_ptr = bias[bias_idx];
    const T in_ptr = data[tid];
    T packed_val;
    packed_val.x = in_ptr.x + bias_ptr.x;
    packed_val.y = in_ptr.y + bias_ptr.y;
    packed_val.z = in_ptr.z + bias_ptr.z;
    packed_val.w = in_ptr.w + bias_ptr.w;
    if (DoRelu) {
      packed_val.x = fmaxf(0.f, packed_val.x);
      packed_val.y = fmaxf(0.f, packed_val.y);
      packed_val.z = fmaxf(0.f, packed_val.z);
      packed_val.w = fmaxf(0.f, packed_val.w);
    }
    data[tid] = packed_val;
  }
}

template <typename T, bool DoRelu, int BlockDim>
__global__ void InplaceAddReluKernel(const int N, const T* bias, T* data) {
  int offset = blockIdx.x * N;

  for (int i = threadIdx.x; i < N; i += BlockDim) {
    T temp;
#if defined(__HIPCC__) || __CUDA_ARCH__ >= 350
    temp = __ldg(data + offset + i) + __ldg(bias + i);
#else
    temp = data[offset + i] + bias[i];
#endif
    if (DoRelu) {
      data[offset + i] = static_cast<int>(temp > 0) * temp;
    } else {
      data[offset + i] = temp;
    }
  }
}

template <bool DoRelu, int BlockDim>
__global__ void InplaceAddReluKernel(const int N,
                                     const float16* bias,
                                     float16* data) {
  int offset = blockIdx.x * N;
  for (int i = threadIdx.x; i < N; i += BlockDim) {
    float16 temp;
    temp = data[offset + i] + bias[i];
    if (DoRelu) {
      data[offset + i] = fmaxf(0.f, temp);
    } else {
      data[offset + i] = temp;
    }
  }
}

template <typename T>
void AddReluKernel(
    gpuStream_t stream, const int M, const int N, T* Y, const T* B, bool relu) {
  if (N % 4 == 0) {
    const int threads = 256;
    const int num = M * N / 4;
    const int blocks = (num + threads - 1) / threads;
    typedef typename FcTypeTraits<T>::Type trans_type;
    auto* bias_ptr_v4 = reinterpret_cast<const trans_type*>(B);
    auto* data_ptr_v4 = reinterpret_cast<trans_type*>(Y);
    if (relu) {
      bias_relu_v4<trans_type, true><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v4, data_ptr_v4, N / 4);
    } else {
      bias_relu_v4<trans_type, false><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v4, data_ptr_v4, N / 4);
    }
  } else {
    const int threads = 256;
    const int blocks = M;

    if (relu) {
      InplaceAddReluKernel<T, true, threads>
          <<<blocks, threads, 0, stream>>>(N, B, Y);
    } else {
      InplaceAddReluKernel<T, false, threads>
          <<<blocks, threads, 0, stream>>>(N, B, Y);
    }
  }
}

template <>
void AddReluKernel(gpuStream_t stream,
                   const int M,
                   const int N,
                   float16* Y,
                   const float16* B,
                   bool relu) {
  if (N % 4 == 0) {
    const int threads = 256;
    const int num = M * N / 4;
    const int blocks = (num + threads - 1) / threads;
    typedef typename FcTypeTraits<float16>::Type trans_type;
    auto* bias_ptr_v4 = reinterpret_cast<const trans_type*>(B);
    auto* data_ptr_v4 = reinterpret_cast<trans_type*>(Y);
    if (relu) {
      bias_relu_v4<trans_type, true><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v4, data_ptr_v4, N / 4);
    } else {
      bias_relu_v4<trans_type, false><<<blocks, threads, 0, stream>>>(
          num, bias_ptr_v4, data_ptr_v4, N / 4);
    }
  } else {
    const int threads = 256;
    const int blocks = M;

    if (relu) {
      InplaceAddReluKernel<true, threads>
          <<<blocks, threads, 0, stream>>>(N, B, Y);
    } else {
      InplaceAddReluKernel<false, threads>
          <<<blocks, threads, 0, stream>>>(N, B, Y);
    }
  }
}

template <typename T, typename DeviceContext>
class FusedGemmEpilogueKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();

    const phi::DenseTensor* x = ctx.Input<phi::DenseTensor>("X");
    const phi::DenseTensor* y = ctx.Input<phi::DenseTensor>("Y");
    const phi::DenseTensor* bias = ctx.Input<phi::DenseTensor>("Bias");
    phi::DenseTensor* out = ctx.Output<phi::DenseTensor>("Out");
    phi::DenseTensor* reserve_space =
        ctx.Output<phi::DenseTensor>("ReserveSpace");

    bool trans_x = ctx.Attr<bool>("trans_x");
    bool trans_y = ctx.Attr<bool>("trans_y");

    std::string activation = ctx.Attr<std::string>("activation");
    dev_ctx.Alloc<T>(out, out->numel() * sizeof(T));
    // (M * K) * (K * N)
    auto x_mat_dims =
        phi::flatten_to_2d(x->dims(), trans_x ? 1 : x->dims().size() - 1);
    int64_t M = trans_x ? x_mat_dims[1] : x_mat_dims[0];
    int64_t K = trans_y ? y->dims()[1] : y->dims()[0];
    int64_t N = trans_y ? y->dims()[0] : y->dims()[1];

    void* reserve_data = reserve_space ? reserve_space->data() : nullptr;
    VLOG(6) << "running FusedGemmEpilogueKernel on DCU";
    auto blas = phi::funcs::GetBlas<DeviceContext, T>(dev_ctx);
    blas.GEMM(CblasNoTrans,
              CblasNoTrans,
              M,
              N,
              K,
              static_cast<T>(1.0),
              x->data<T>(),
              y->data<T>(),
              static_cast<T>(0.0),
              out->data<T>());

    auto B = bias->data<T>();
    AddReluKernel(dev_ctx.stream(),
                  static_cast<int>(M),
                  static_cast<int>(N),
                  out->data<T>(),
                  B,
                  false);
  }
};

template <typename T, typename DeviceContext>
class FusedGemmEpilogueGradKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
    auto& dev_ctx = ctx.template device_context<phi::GPUContext>();

    const phi::DenseTensor* dout = ctx.Input<phi::DenseTensor>("DOut");
    const phi::DenseTensor* x = ctx.Input<phi::DenseTensor>("X");
    const phi::DenseTensor* y = ctx.Input<phi::DenseTensor>("Y");
    const phi::DenseTensor* reserve_space =
        ctx.Input<phi::DenseTensor>("ReserveSpace");
    phi::DenseTensor* dx = ctx.Output<phi::DenseTensor>("DX");
    phi::DenseTensor* dy = ctx.Output<phi::DenseTensor>("DY");
    phi::DenseTensor* dbias = ctx.Output<phi::DenseTensor>("DBias");

    std::string activation_grad = ctx.Attr<std::string>("activation_grad");
    bool trans_x = ctx.Attr<bool>("trans_x");
    bool trans_y = ctx.Attr<bool>("trans_y");

    // (M * K) * (K * N)
    auto x_mat_dims =
        phi::flatten_to_2d(x->dims(), trans_x ? 1 : x->dims().size() - 1);
    int64_t M = trans_x ? x_mat_dims[1] : x_mat_dims[0];
    int64_t K = trans_y ? y->dims()[1] : y->dims()[0];
    int64_t N = trans_y ? y->dims()[0] : y->dims()[1];

    VLOG(6) << "Running FusedGemmEpilogueKernel backward on DCU";

    auto blas = phi::funcs::GetBlas<DeviceContext, T>(dev_ctx);
    // fc grad
    if (dx) {
      // dx = matmul(dz, y, None, False, True)
      dev_ctx.Alloc<T>(dx);
      blas.GEMM(CblasNoTrans,
                CblasTrans,
                M,
                K,
                N,
                static_cast<T>(1.0),
                dout->data<T>(),
                y->data<T>(),
                static_cast<T>(0.0),
                dx->data<T>());
    }
    if (dy) {
      dev_ctx.Alloc<T>(dy);
      // dy = matmul(x, dz, None, True, False)
      blas.GEMM(CblasTrans,
                CblasNoTrans,
                K,
                N,
                M,
                static_cast<T>(1.0),
                x->data<T>(),
                dout->data<T>(),
                static_cast<T>(0.0),
                dy->data<T>());
    }
    // bas grad
    if (dbias) {
      dev_ctx.Alloc<T>(dbias);
      //  dbias = np.sum(dz, axis=0, keepdims=False)
      const std::vector<int64_t> reduce_dims{0};
      phi::Reduce<T, phi::kps::AddFunctor, phi::kps::IdentityFunctor>(
          dev_ctx, *dout, false, reduce_dims, false, dout->dtype(), dbias);
    }
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;
PD_REGISTER_STRUCT_KERNEL(fused_gemm_epilogue,
                          GPU,
                          ALL_LAYOUT,
                          ops::FusedGemmEpilogueKernel,
                          float,
                          double,
                          plat::float16) {}

PD_REGISTER_STRUCT_KERNEL(fused_gemm_epilogue_grad,
                          GPU,
                          ALL_LAYOUT,
                          ops::FusedGemmEpilogueGradKernel,
                          float,
                          double,
                          plat::float16) {}
